#include "hip/hip_runtime.h"
#include "cuda_median_filter.h"
#include <hip/hip_runtime.h>
#include <algorithm>
#include <map>

// 显存缓冲池结构
struct BufferPool {
    unsigned char* d_src;
    unsigned char* d_dst;
    int allocated_width;
    int allocated_height;
    bool is_allocated;
};

// 全局缓冲池映射
static std::map<CudaBufferPool, BufferPool> g_buffer_pools;

// CUDA核函数：中值滤波
__global__ void median_filter_kernel(const unsigned char src, unsigned char dst, int width, int height, int kernel_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int half = kernel_size / 2;
    if (x >= width || y >= height) return;

    unsigned char window[49]; // 最大支持7x7
    int count = 0;
    for (int dy = -half; dy <= half; ++dy) {
        for (int dx = -half; dx <= half; ++dx) {
            int ix = min(max(x + dx, 0), width - 1);
            int iy = min(max(y + dy, 0), height - 1);
            window[count++] = src[iy * width + ix];
        }
    }
  // 排序取中值
    for (int i = 0; i < count - 1; ++i) {
        for (int j = 0; j < count - i - 1; ++j) {
            if (window[j] > window[j + 1]) {
                unsigned char tmp = window[j];
                window[j] = window[j + 1];
                window[j + 1] = tmp;
            }
        }
    }
    dst[y * width + x] = window[count / 2];
}

// 初始化显存缓冲池
extern "C" DLL_EXPORT 
int CudaBufferPool cuda_init_buffer_pool(int max_width, int max_height)
{
    BufferPool* pool = new BufferPool();
    pool->d_src = nullptr;
    pool->d_dst = nullptr;
    pool->allocated_width = 0;
    pool->allocated_height = 0;
    pool->is_allocated = false;
    
    g_buffer_pools[pool] = *pool;
    return pool;
}

// 清理显存缓冲池
extern "C" DLL_EXPORT
int cuda_cleanup_buffer_pool(CudaBufferPool pool)
{
    if (!pool) return -1;
    
    auto it = g_buffer_pools.find(pool);
    if (it == g_buffer_pools.end()) return -2;
    
    BufferPool& buffer_pool = it->second;
    if (buffer_pool.is_allocated) {
        hipFree(buffer_pool.d_src);
        hipFree(buffer_pool.d_dst);
    }
    
    g_buffer_pools.erase(it);
    delete (BufferPool*)pool;
    return 0;
}

// 使用缓冲池的中值滤波（高性能版本）
extern "C" DLL_EXPORT
int cuda_median_filter_with_pool(
    CudaBufferPool pool,
    const unsigned char* src,
    int width,
    int height,
    int kernel_size,
    unsigned char* dst)
{
    if (!pool || !src || !dst || width <= 0 || height <= 0 || 
        kernel_size < 3 || (kernel_size % 2) == 0 || kernel_size > 7)
        return -1;
    
    auto it = g_buffer_pools.find(pool);
    if (it == g_buffer_pools.end()) return -2;
    
    BufferPool& buffer_pool = it->second;
    size_t img_size = width  height  sizeof(unsigned char);
    
    // 检查是否需要重新分配显存
    if (!buffer_pool.is_allocated || 
        buffer_pool.allocated_width < width || 
        buffer_pool.allocated_height < height) {
        
        // 释放旧的显存
        if (buffer_pool.is_allocated) {
            hipFree(buffer_pool.d_src);
            hipFree(buffer_pool.d_dst);
        }
        
        // 分配新的显存
        hipError_t err = hipMalloc(&buffer_pool.d_src, img_size);
        if (err != hipSuccess) return -3;
        
        err = hipMalloc(&buffer_pool.d_dst, img_size);
        if (err != hipSuccess) {
            hipFree(buffer_pool.d_src);
            return -4;
        }
        
        buffer_pool.allocated_width = width;
        buffer_pool.allocated_height = height;
        buffer_pool.is_allocated = true;
    }
    
    // 复制数据到GPU
    hipError_t err = hipMemcpy(buffer_pool.d_src, src, img_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) return -5;
    
    // 执行滤波
    dim3 block(16, 16);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);
    median_filter_kernel<<<grid, block>>>(buffer_pool.d_src, buffer_pool.d_dst, width, height, kernel_size);
    
    err = hipDeviceSynchronize();
    if (err != hipSuccess) return -6;
    
    // 复制结果回CPU
    err = hipMemcpy(dst, buffer_pool.d_dst, img_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) return -7;
    
    return 0;
}

// 原始接口（保持兼容性）
extern "C" DLL_EXPORT
int cuda_median_filter(const unsigned char src, int width, int height, int kernel_size, unsigned char dst)
{
    if (!src || !dst || width <= 0 || height <= 0 || kernel_size < 3 || (kernel_size % 2) == 0 || kernel_size > 7)
        return -1;
    size_t img_size = width  height  sizeof(unsigned char);
    unsigned char d_src = nullptr, d_dst = nullptr;
    hipError_t err;
    err = hipMalloc(&d_src, img_size);
    if (err != hipSuccess) return -2;
    err = hipMalloc(&d_dst, img_size);
    if (err != hipSuccess) { hipFree(d_src); return -3; }
    err = hipMemcpy(d_src, src, img_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) { hipFree(d_src); hipFree(d_dst); return -4; }

    dim3 block(16, 16);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);
    median_filter_kernel<<<grid, block>>>(d_src, d_dst, width, height, kernel_size);
    err = hipDeviceSynchronize();
    if (err != hipSuccess) { hipFree(d_src); hipFree(d_dst); return -5; }
    err = hipMemcpy(dst, d_dst, img_size, hipMemcpyDeviceToHost);
    hipFree(d_src);
    hipFree(d_dst);
    if (err != hipSuccess) return -6;
    return 0;
}
